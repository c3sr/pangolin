#include "hip/hip_runtime.h"
 #include "graph/cudamemcpy_tc.hpp"
 #include "graph/logger.hpp"
 #include "graph/utilities.hpp"

 #include "graph/dag2019.hpp"

const int BLOCK_DIM_X = 128;

__global__ static void kernel_tc(size_t * __restrict__ triangleCounts, const Int *edgeSrc, const Int *edgeDst, const Int *nodes, const size_t edgeOffset, const size_t numEdges){
     
    const Int gx = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
    
    for (Int i = gx + edgeOffset; i < edgeOffset + numEdges; i += BLOCK_DIM_X * gridDim.x) {

        const Int src = edgeSrc[i];
        const Int dst = edgeDst[i];

        Int src_edge = nodes[src];
        const Int src_edge_end = nodes[src + 1];

        Int dst_edge = nodes[dst];
        const Int dst_edge_end = nodes[dst + 1];


        size_t count = 0;

        bool update_u = true;
        bool update_v = true;
        while (src_edge < src_edge_end && dst_edge < dst_edge_end){
            Int u, v;
            if (update_u) u = edgeDst[src_edge];
            if (update_v) v = edgeDst[dst_edge];

            // the two nodes that make up this edge both have a common dst
            if (u == v) {
                ++count;
                ++src_edge;
                ++dst_edge;
                update_u = true;
                update_v = true;
            }
            else if (u < v){
                ++src_edge;
                update_u = true;
                update_v = false;
            }
            else {
                ++dst_edge;
                update_u = false;
                update_v = true;
            }
        }

        triangleCounts[i] = count;
    }
}

CudaMemcpyTC::CudaMemcpyTC() {
    LOG(debug, "ctor GPU triangle counter, sizeof(Int) = {}", sizeof(Int));

    int numDev;
    CUDA_RUNTIME(hipGetDeviceCount(&numDev));
    for (int i = 0; i < numDev; ++i) {
        LOG(info, "Initializing CUDA device {}", i);
        CUDA_RUNTIME(hipSetDevice(i));
        CUDA_RUNTIME(hipFree(0));
    }


    }

CudaMemcpyTC::~CudaMemcpyTC() {
    LOG(debug, "dtor GPU triangle counter");
    LOG(debug, "unregistering/freeing CUDA memory");
    CUDA_RUNTIME(hipFree(edgeSrc_d_));
    CUDA_RUNTIME(hipFree(edgeDst_d_));
    CUDA_RUNTIME(hipFree(nodes_d_));
    CUDA_RUNTIME(hipHostFree(triangleCounts_));
}

void CudaMemcpyTC::read_data(const std::string &path) {

    LOG(info, "reading {}", path);
    auto edgeList = EdgeList::read_tsv(path);
    LOG(debug, "building DAG");
    hostDAG_ = DAG2019::from_edgelist(edgeList);

    LOG(info, "{} nodes", hostDAG_.num_nodes());
    LOG(info, "{} edges", hostDAG_.num_edges());
}

void CudaMemcpyTC::setup_data() {
    const size_t edgeBytes = hostDAG_.edgeSrc_.size() * sizeof(Int);
    const size_t nodeBytes = hostDAG_.nodes_.size() * sizeof(Int);
    const size_t countBytes = hostDAG_.num_edges() * sizeof(*triangleCounts_);

    CUDA_RUNTIME(hipMalloc(&edgeSrc_d_, edgeBytes));
    CUDA_RUNTIME(hipMalloc(&edgeDst_d_, edgeBytes));
    CUDA_RUNTIME(hipMalloc(&nodes_d_, nodeBytes));
    CUDA_RUNTIME(hipHostAlloc(&triangleCounts_, countBytes, hipHostMallocMapped));

    CUDA_RUNTIME(hipMemcpy(edgeSrc_d_, hostDAG_.edgeSrc_.data(), edgeBytes, hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpy(edgeDst_d_, hostDAG_.edgeDst_.data(), edgeBytes, hipMemcpyDefault)); 
    CUDA_RUNTIME(hipMemcpy(nodes_d_, hostDAG_.nodes_.data(), nodeBytes, hipMemcpyDefault));

}

size_t CudaMemcpyTC::count() {
    
    int numDev;
    CUDA_RUNTIME(hipGetDeviceCount(&numDev));

    // split edges into devices
    size_t edgesPerDevice = (hostDAG_.num_edges() + numDev - 1) / numDev;
    LOG(debug, "{} edges per GPU", edgesPerDevice);

    size_t edgeOffset = 0;
    for (int i = 0; i < numDev; ++i) {
        CUDA_RUNTIME(hipSetDevice(i));


        size_t edgeCount = std::min(edgesPerDevice, hostDAG_.num_edges() - edgeOffset);
        LOG(debug, "GPU {} edges {}+{}", i, edgeOffset, edgeCount);


        dim3 dimBlock(BLOCK_DIM_X);
        dim3 dimGrid((hostDAG_.num_edges() + dimBlock.x - 1) / dimBlock.x);
    
        LOG(debug, "kernel dims {} x {}", dimGrid.x, dimBlock.x);
        kernel_tc<<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_, nodes_d_, edgeOffset, edgeCount);
        edgeOffset += edgesPerDevice;
    }
    
    for (int i = 0; i < numDev; ++i) {
        CUDA_RUNTIME(hipSetDevice(i));
        LOG(debug, "Waiting for GPU {}", i);
        CUDA_RUNTIME(hipDeviceSynchronize());
    }

    auto start = std::chrono::system_clock::now();
    size_t total = 0;
    for(size_t i = 0; i < hostDAG_.num_edges(); ++i) {
        total += triangleCounts_[i];
    }
    auto elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(debug, "CPU reduction {}s", elapsed);


    return total;
}
