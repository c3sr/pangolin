#include "hip/hip_runtime.h"
#include "graph/um_tc.hpp"
#include "graph/logger.hpp"
#include "graph/utilities.hpp"
#include "graph/reader/gc_tsv_reader.hpp"
#include "graph/dag2019.hpp"

#include <set>
#include <nvToolsExt.h>

__global__ static void kernel_tc(size_t *triangleCounts, const Int *edgeSrc, const Int *edgeDst, const Int *nodes, const size_t edgeOffset, const size_t numEdges){
     
    const Int gx = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (Int i = gx + edgeOffset; i < edgeOffset + numEdges; i += blockDim.x * gridDim.x) {

        // get the src and dst node for this edge
        const Int src = edgeSrc[i];
        const Int dst = edgeDst[i];

        Int src_edge = nodes[src];
        const Int src_edge_end = nodes[src + 1];

        Int dst_edge = nodes[dst];
        const Int dst_edge_end = nodes[dst + 1];

        size_t count = 0;

        bool readSrc = true;
        bool readDst = true;

        while (src_edge < src_edge_end && dst_edge < dst_edge_end){

            Int u, v;

            if (readSrc) {
                u = edgeDst[src_edge];
                readSrc = false;
            }

            if (readDst) {
                v = edgeDst[dst_edge];
                readDst = false;
            }

            // the two nodes that make up this edge both have a common dst
            if (u == v) {
                ++count;
                ++src_edge;
                ++dst_edge;
                readSrc = true;
                readDst = true;
            }
            else if (u < v){
                ++src_edge;
                readSrc = true;
            }
            else {
                ++dst_edge;
                readDst = true;
            }
        }

        triangleCounts[i] = count;
    }
}



UMTC::UMTC(Config &c) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(debug, "ctor GPU triangle counter, sizeof(Int) = {}", sizeof(Int));

    gpus_ = c.gpus_;

    if (gpus_.empty()) {
        LOG(critical, "Unified-memory edge-set intersection triangle counter requires >= 1 GPU");
        exit(-1);
    }

    for (int dev : std::set<int>(gpus_.begin(), gpus_.end())) {
        LOG(info, "Initializing CUDA device {}", dev);
        CUDA_RUNTIME(hipSetDevice(dev));
        CUDA_RUNTIME(hipFree(0));
    }
    nvtxRangePop();
}

UMTC::~UMTC() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    CUDA_RUNTIME(hipFree(edgeSrc_d_));
    CUDA_RUNTIME(hipFree(edgeDst_d_));
    CUDA_RUNTIME(hipFree(nodes_d_));
    CUDA_RUNTIME(hipFree(triangleCounts_));
    nvtxRangePop();
}

void UMTC::read_data(const std::string &path) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(info, "reading {}", path);
    GraphChallengeTSVReader reader(path);
    auto edgeList = reader.read_edges();
    LOG(debug, "building DAG");
    hostDAG_ = DAG2019::from_edgelist(edgeList);

    LOG(info, "{} nodes", hostDAG_.num_nodes());
    LOG(info, "{} edges", hostDAG_.num_edges());
    nvtxRangePop();
}

void UMTC::setup_data() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t edgeBytes = hostDAG_.edgeSrc_.size() * sizeof(Int);
    const size_t nodeBytes = hostDAG_.nodes_.size() * sizeof(Int);
    const size_t countBytes = hostDAG_.num_edges() * sizeof(*triangleCounts_);

    LOG(debug, "allocating unified memory");
    CUDA_RUNTIME(hipMallocManaged(&edgeSrc_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&edgeDst_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&nodes_d_, nodeBytes));
    CUDA_RUNTIME(hipMallocManaged(&triangleCounts_, countBytes));

    LOG(debug, "copying to unified memory");
    CUDA_RUNTIME(hipMemcpy(edgeSrc_d_, hostDAG_.edgeSrc_.data(), edgeBytes, hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpy(edgeDst_d_, hostDAG_.edgeDst_.data(), edgeBytes, hipMemcpyDefault)); 
    CUDA_RUNTIME(hipMemcpy(nodes_d_, hostDAG_.nodes_.data(), nodeBytes, hipMemcpyDefault));
    nvtxRangePop();
}

size_t UMTC::count() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t numDev = gpus_.size();

    // split edges among devices
    size_t edgesPerDevice = (hostDAG_.num_edges() + numDev - 1) / numDev;
    LOG(debug, "{} edges per GPU", edgesPerDevice);

    size_t edgeOffset = 0;
    for (int i : gpus_) {
        CUDA_RUNTIME(hipSetDevice(i));

        size_t edgeCount = std::min(edgesPerDevice, hostDAG_.num_edges() - edgeOffset);
        LOG(debug, "GPU {} edges {}+{}", i, edgeOffset, edgeCount);


        dim3 dimBlock(512);
        dim3 dimGrid((edgeCount + dimBlock.x - 1) / dimBlock.x);
    
        LOG(debug, "kernel dims {} x {}", dimGrid.x, dimBlock.x);
        kernel_tc<<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_, nodes_d_, edgeOffset, edgeCount);
        edgeOffset += edgesPerDevice;
    }
    
    for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
        CUDA_RUNTIME(hipSetDevice(i));
        LOG(debug, "Waiting for GPU {}", i);
        CUDA_RUNTIME(hipDeviceSynchronize());
    }

    nvtxRangePush("final reduction");
    auto start = std::chrono::system_clock::now();
    size_t total = 0;
    for(size_t i = 0; i < hostDAG_.num_edges(); ++i) {
        total += triangleCounts_[i];
    }
    auto elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(debug, "CPU reduction {}s", elapsed);

    nvtxRangePop();
    return total;
}
