#include "hip/hip_runtime.h"
#include "graph/um_tc.hpp"
#include "graph/logger.hpp"
#include "graph/utilities.hpp"
#include "graph/reader/gc_tsv_reader.hpp"
#include "graph/dag2019.hpp"

#include <set>
#include <nvToolsExt.h>
#include <hipcub/hipcub.hpp>

__device__ static bool binary_search(const Int* const array, Int left, Int right, const Int search_val) {
    while(left <= right) {
        int mid = (left + right)/2;
        int val = array[mid];
        if(val < search_val) {
            left = mid + 1;
        } else if(val > search_val) {
            right = mid - 1;
        } else { // val == search_val
            return 1;
        }
    }
    return 0;
}

__device__ static size_t intersection_count(const Int *const aBegin, const Int *const aEnd, const Int *const bBegin, const Int *const bEnd) {
    size_t count = 0;
    const Int *ap = aBegin;
    const Int *bp = bBegin;

    while (ap < aEnd && bp < bEnd) {

        if (*ap == *bp) {
            ++count;
            ++ap;
            ++bp;
        }
        else if (*ap < *bp){
            ++ap;
        }
        else {
            ++bp;
        }
    }
    return count;
}

template <size_t BLOCK_DIM_X>
__global__ static void kernel_tc(size_t * __restrict__ triangleCounts, const Int *edgeSrc, const Int *edgeDst, const Int *nodes, const size_t edgeOffset, const size_t numEdges){
     
    const Int gx = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
    
    for (Int i = gx + edgeOffset; i < edgeOffset + numEdges; i += BLOCK_DIM_X * gridDim.x) {

        // get the src and dst node for this edge
        const Int src = edgeSrc[i];
        const Int dst = edgeDst[i];

        Int src_edge = nodes[src];
        const Int src_edge_end = nodes[src + 1];

        Int dst_edge = nodes[dst];
        const Int dst_edge_end = nodes[dst + 1];

        size_t count = 0;



            count = intersection_count(&edgeDst[src_edge], &edgeDst[src_edge_end], &edgeDst[dst_edge], &edgeDst[dst_edge_end]);



        /*
        bool readSrc = true;
        bool readDst = true;
        while (src_edge < src_edge_end && dst_edge < dst_edge_end) {

            Int u, v;

            if (readSrc) {
                u = edgeDst[src_edge];
                readSrc = false;
            }

            if (readDst) {
                v = edgeDst[dst_edge];
                readDst = false;
            }

            // the two nodes that make up this edge both have a common dst
            if (u == v) {
                ++count;
                ++src_edge;
                ++dst_edge;
                readSrc = true;
                readDst = true;
            }
            else if (u < v){
                ++src_edge;
                readSrc = true;
            }
            else {
                ++dst_edge;
                readDst = true;
            }
        }
        */

        triangleCounts[i] = count;
    }
}


template <size_t BLOCK_DIM_X>
__global__ static void kernel_tc2(size_t * __restrict__ triangleCounts, const Int *edgeSrc, const Int *edgeDst, const Int *nodes, const size_t edgeOffset, const size_t numEdges){

    static_assert(BLOCK_DIM_X > 0, "threadblock should have at least 1 thread");
    static_assert(BLOCK_DIM_X % 32 == 0, "require BLOCK_DIM_X to be an integer number of warps");
    const Int WARPS_PER_BLOCK = BLOCK_DIM_X / 32;

    const Int gx = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
    const Int gwx = gx / 32;
    const Int lx = gx % 32;
    
    typedef hipcub::WarpReduce<size_t> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS_PER_BLOCK];

    // each edge gets a warp
    for (Int i = gwx + edgeOffset; i < edgeOffset + numEdges; i += WARPS_PER_BLOCK * gridDim.x) {

        // get the src and dst node for this edge
        #if 0
        Int src_edge, src_edge_end, dst_edge, dst_edge_end;
        if (lx == 0) {
            Int src = edgeSrc[i];
            Int dst = edgeDst[i];
            src_edge = nodes[src];
            src_edge_end = nodes[src + 1];
            dst_edge = nodes[dst];
            dst_edge_end = nodes[dst + 1];
        }
        src_edge = hipcub::ShuffleIndex<Int>(src_edge, 0, 32, 0xffffffff);
        dst_edge = hipcub::ShuffleIndex<Int>(dst_edge, 0, 32, 0xffffffff);
        src_edge_end = hipcub::ShuffleIndex<Int>(src_edge_end, 0, 32, 0xffffffff);
        dst_edge_end = hipcub::ShuffleIndex<Int>(dst_edge_end, 0, 32, 0xffffffff);
        #else

        // get the src and dst node for this edge
        const Int src = edgeSrc[i];
        const Int dst = edgeDst[i];
        const Int src_edge = nodes[src];
        const Int src_edge_end = nodes[src + 1];
        const Int dst_edge = nodes[dst];
        const Int dst_edge_end = nodes[dst + 1];
        #endif


        size_t count = 0;

        // binary search of larger list
        if (src_edge_end - src_edge < dst_edge_end - dst_edge) {
            for (const Int *u = &edgeDst[src_edge] + lx; u < &edgeDst[src_edge_end]; u += 32) {
                count += binary_search(edgeDst, dst_edge, dst_edge_end-1, *u);
            }
        } else {
            for (const Int *u = &edgeDst[dst_edge] + lx; u < &edgeDst[dst_edge_end]; u += 32) {
                count += binary_search(edgeDst, src_edge, src_edge_end-1, *u);
            }                
        }


        // Obtain one input item per thread
        // Return the warp-wide sums to each lane0 (threads 0, 32, 64, and 96)
        int warp_id = threadIdx.x / 32;
        size_t aggregate = WarpReduce(temp_storage[warp_id]).Sum(count);

        if (lx == 0) {
            triangleCounts[i] = aggregate;
        }
    }
}


UMTC::UMTC(Config &c) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(debug, "ctor GPU triangle counter, sizeof(Int) = {}", sizeof(Int));

    gpus_ = c.gpus_;

    if (gpus_.empty()) {
        LOG(critical, "Unified-memory edge-set intersection triangle counter requires >= 1 GPU");
        exit(-1);
    }

    for (int dev : std::set<int>(gpus_.begin(), gpus_.end())) {
        LOG(info, "Initializing CUDA device {}", dev);
        CUDA_RUNTIME(hipSetDevice(dev));
        CUDA_RUNTIME(hipFree(0));
        if (0 == cudaDeviceProps_.count(dev)) {
            CUDA_RUNTIME(hipGetDeviceProperties(&cudaDeviceProps_[dev], dev));
        }
    }
    nvtxRangePop();
}

UMTC::~UMTC() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    CUDA_RUNTIME(hipFree(edgeSrc_d_));
    CUDA_RUNTIME(hipFree(edgeDst_d_));
    CUDA_RUNTIME(hipFree(nodes_d_));
    CUDA_RUNTIME(hipFree(triangleCounts_));
    nvtxRangePop();
}

void UMTC::read_data(const std::string &path) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(info, "reading {}", path);
    GraphChallengeTSVReader reader(path);
    auto edgeList = reader.read_edges();
    LOG(debug, "building DAG");
    hostDAG_ = DAG2019::from_edgelist(edgeList);

    LOG(info, "{} nodes", hostDAG_.num_nodes());
    LOG(info, "{} edges", hostDAG_.num_edges());
    nvtxRangePop();
}

void UMTC::setup_data() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t edgeBytes = hostDAG_.edgeSrc_.size() * sizeof(Int);
    const size_t nodeBytes = hostDAG_.nodes_.size() * sizeof(Int);
    const size_t countBytes = hostDAG_.num_edges() * sizeof(*triangleCounts_);

    LOG(debug, "allocating unified memory");
    CUDA_RUNTIME(hipMallocManaged(&edgeSrc_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&edgeDst_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&nodes_d_, nodeBytes));
    CUDA_RUNTIME(hipMallocManaged(&triangleCounts_, countBytes));

    LOG(debug, "copying to unified memory");
    CUDA_RUNTIME(hipMemcpy(edgeSrc_d_, hostDAG_.edgeSrc_.data(), edgeBytes, hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpy(edgeDst_d_, hostDAG_.edgeDst_.data(), edgeBytes, hipMemcpyDefault)); 
    CUDA_RUNTIME(hipMemcpy(nodes_d_, hostDAG_.nodes_.data(), nodeBytes, hipMemcpyDefault));

    // processor id is ignored
    CUDA_RUNTIME(hipMemAdvise(edgeSrc_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
    CUDA_RUNTIME(hipMemAdvise(edgeDst_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
    CUDA_RUNTIME(hipMemAdvise(nodes_d_, nodeBytes, hipMemAdviseSetReadMostly, 0));
    for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
      if (cudaDeviceProps_[i].concurrentManagedAccess) {
        CUDA_RUNTIME(hipMemAdvise(edgeSrc_d_, edgeBytes, hipMemAdviseSetAccessedBy, i));
        CUDA_RUNTIME(hipMemAdvise(edgeDst_d_, edgeBytes, hipMemAdviseSetAccessedBy, i));
        CUDA_RUNTIME(hipMemAdvise(nodes_d_, nodeBytes, hipMemAdviseSetAccessedBy, i));
      } else {
          LOG(warn, "skipping hipMemAdviseSetAccessedBy for device {}: hipDeviceProp_t.concurrentManagedAccess = 0", i);
      }
    }
    nvtxRangePop();
}

size_t UMTC::count() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t numDev = gpus_.size();

    // split edges among devices
    size_t edgesPerDevice = (hostDAG_.num_edges() + numDev - 1) / numDev;
    LOG(debug, "{} edges per GPU", edgesPerDevice);

    size_t edgeOffset = 0;
    for (int i : gpus_) {
        CUDA_RUNTIME(hipSetDevice(i));

        size_t edgeCount = std::min(edgesPerDevice, hostDAG_.num_edges() - edgeOffset);
        LOG(debug, "GPU {} edges {}+{}", i, edgeOffset, edgeCount);

        const size_t BLOCK_SIZE = 128;
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid((edgeCount + dimBlock.x - 1) / dimBlock.x);
    
        LOG(debug, "kernel dims {} x {}", dimGrid.x, dimBlock.x);
        kernel_tc2<BLOCK_SIZE><<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_, nodes_d_, edgeOffset, edgeCount);
        edgeOffset += edgesPerDevice;
    }
    
    for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
        CUDA_RUNTIME(hipSetDevice(i));
        LOG(debug, "Waiting for GPU {}", i);
        CUDA_RUNTIME(hipDeviceSynchronize());
    }

    nvtxRangePush("final reduction");
    auto start = std::chrono::system_clock::now();
    size_t total = 0;
    for(size_t i = 0; i < hostDAG_.num_edges(); ++i) {
        total += triangleCounts_[i];
    }
    auto elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(debug, "CPU reduction {}s", elapsed);

    nvtxRangePop();
    return total;
}
