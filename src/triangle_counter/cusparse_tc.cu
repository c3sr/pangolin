#include <memory>
#include <cmath>
#include <nvToolsExt.h>
#include <hipcub/hipcub.hpp>

#include "pangolin/logger.hpp"
#include "pangolin/triangle_counter/cusparse_tc.hpp"
#include "pangolin/reader/edge_list_reader.hpp"
#include "pangolin/utilities.hpp"
#include "pangolin/cusparse.hpp"
#include "pangolin/narrow.hpp"
#include "pangolin/algorithm/elementwise.cuh"
#include "pangolin/algorithm/fill.cuh"

PANGOLIN_BEGIN_NAMESPACE()



CusparseTC::CusparseTC(Config &c) : descrA_(nullptr), descrC_(nullptr)
{

    if (c.gpus_.size() == 0)
    {
        LOG(critical, "CusparseTC requires 1 GPU");
        exit(-1);
    }
    
    gpu_ = c.gpus_[0];
    if (c.gpus_.size() > 1)
    {
        LOG(warn, "CusparseTC requires exactly 1 GPU. Selected GPU {}", gpu_);
    }

    LOG(debug, "create CUSparse handle");
    CUSPARSE(hipsparseCreate(&handle_));

    int version;
    CUSPARSE(hipsparseGetVersion(handle_, &version));
    LOG(info, "CUSparse version {}", version);

    CUSPARSE(hipsparseCreateMatDescr(&descrA_));
    CUSPARSE(hipsparseCreateMatDescr(&descrC_));
}



void CusparseTC::read_data(const std::string &path)
{
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(info, "reading {}", path);
    auto *reader = pangolin::EdgeListReader::from_file(path);
    auto edgeList = reader->read_all();
    if (edgeList.size() == 0) {
        LOG(warn, "empty edge list");
    }
    LOG(debug, "building A");
    A_ = GPUCSR<int>::from_edgelist(edgeList, [](const Edge &e) {
        return e.second >= e.first; // keep src > dst
    });
    nvtxRangePop();
}



void CusparseTC::setup_data()
{
    assert(sizeof(Int) == sizeof(int));
}



size_t CusparseTC::count()
{

    const int m = checked_narrow(A_.num_rows());
    const int n = checked_narrow(A_.max_col()+1);
    const int k = checked_narrow(A_.max_col()+1);
    LOG(debug, "CUSparse product m={} n={} k={}", m, n, k);

    const hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    const int nnzA = checked_narrow(A_.nnz());
    const int *csrRowPtrA = A_.deviceRowPtr();
    const int *csrColIndA = A_.deviceColInd();
    assert(nnzA == csrRowPtrA[m] - csrRowPtrA[0]);
    LOG(debug, "A has {} nonzeros", nnzA);

    int *csrRowPtrC = nullptr;
    LOG(debug, "allocate {} rows for C", m);
    CUDA_RUNTIME(hipMallocManaged(&csrRowPtrC, sizeof(int) * (m + 1)));
    
    LOG(debug, "compute C nnzs");
    int nnzC;
    int *nnzTotalDevHostPtr = &nnzC;
    CUSPARSE(hipsparseSetPointerMode(handle_, HIPSPARSE_POINTER_MODE_HOST));
    CUSPARSE(hipsparseXcsrgemmNnz(handle_, transA, transA, m, n, k, 
        descrA_, nnzA, csrRowPtrA, csrColIndA,
        descrA_, nnzA, csrRowPtrA, csrColIndA,
        descrC_, csrRowPtrC, nnzTotalDevHostPtr)
    );
    if (nullptr != nnzTotalDevHostPtr){
        TRACE("get nnzC from nnzTotalDevHostPtr");
        nnzC = *nnzTotalDevHostPtr;
        assert(nnzC == csrRowPtrC[m] - csrRowPtrC[0]);
    } else {
        int baseC;
        nnzC = csrRowPtrC[m];
        baseC = csrRowPtrC[0];
        nnzC -= baseC;
    }
    LOG(debug, "C has {} nonzeros", nnzC);

    int *csrColIndC = nullptr;
    float *csrValC = nullptr;
    LOG(debug, "allocate {} B for csrColIndC", sizeof(int) * nnzC);
    CUDA_RUNTIME(hipMallocManaged(&csrColIndC, sizeof(int) * nnzC));
    LOG(debug, "allocate {} B for csrValC", sizeof(float) * nnzC);
    CUDA_RUNTIME(hipMallocManaged(&csrValC, sizeof(float) * nnzC));


    float *csrValA = nullptr;
    LOG(debug, "allocate/fill {} B for A csrValA", sizeof(float) * nnzA);
    CUDA_RUNTIME(hipMallocManaged(&csrValA, sizeof(float) * nnzA));
    pangolin::device_fill(csrValA, nnzA, 1.0f);
    CUDA_RUNTIME(hipDeviceSynchronize());

    LOG(debug, "hipsparseScsrgemm");
    CUSPARSE(hipsparseScsrgemm(handle_, transA, transA, m, n, k,
        descrA_, nnzA,
        csrValA, csrRowPtrA, csrColIndA,
        descrA_, nnzA,
        csrValA, csrRowPtrA, csrColIndA,
        descrC_,
        csrValC, csrRowPtrC, csrColIndC
    ));

    LOG(debug, "hadamard product");
    // c .*= A
    constexpr size_t dimBlockX = 256;
    const size_t dimGridX = (m + dimBlockX - 1) / dimBlockX;
    
    pangolin::csr_elementwise_inplace<dimBlockX><<<dimGridX, dimBlockX>>>(
        csrRowPtrC,
        csrColIndC,
        csrValC,
        csrRowPtrA,
        csrColIndA,
        csrValA,
        m
    );
    CUDA_RUNTIME(hipGetLastError());

    float *deviceTotal;
    CUDA_RUNTIME(hipMallocManaged(&deviceTotal, sizeof(*deviceTotal)));
    *deviceTotal = 0;

    // Reduce the final non-zeros
    void     *d_temp_storage = nullptr;
    size_t   temp_storage_bytes = 0;
    LOG(debug, "compute reduction storage requirements");
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, csrValC, deviceTotal, nnzC);
    LOG(debug, "allocate {} B for temporary reduction storage", temp_storage_bytes);
    CUDA_RUNTIME(hipMalloc(&d_temp_storage, temp_storage_bytes));
    LOG(debug, "device reduction");
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, csrValC, deviceTotal, nnzC);
    TRACE("free temporary reduction storage", temp_storage_bytes);
    CUDA_RUNTIME(hipFree(d_temp_storage));
    
    uint64_t total = *deviceTotal;
    LOG(debug, "total is {}", total);

    CUDA_RUNTIME(hipFree(deviceTotal));
    return total;
}



CusparseTC::~CusparseTC() {
    LOG(debug, "destroy A");
    CUSPARSE(hipsparseDestroyMatDescr(descrA_));
    LOG(debug, "destroy C");
    CUSPARSE(hipsparseDestroyMatDescr(descrC_));
    LOG(debug, "destroy handle");
    CUSPARSE(hipsparseDestroy(handle_));
}

PANGOLIN_END_NAMESPACE()