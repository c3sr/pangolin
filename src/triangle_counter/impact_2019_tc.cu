#include "hip/hip_runtime.h"
/// \file

#include "pangolin/triangle_counter/impact_2019_tc.hpp"
#include "pangolin/logger.hpp"
#include "pangolin/utilities.hpp"
#include "pangolin/reader/gc_tsv_reader.hpp"

#include <nvToolsExt.h>
#include <limits>
#include <hipcub/hipcub.hpp>

__device__ static size_t linear_intersection_count(const Int *const aBegin, const Int *const aEnd, const Int *const bBegin, const Int *const bEnd) {
    size_t count = 0;
    const Int *ap = aBegin;
    const Int *bp = bBegin;

    if (ap < aEnd && bp < bEnd) {

        bool loadA = false;
        bool loadB = false;
        Int a = *ap;
        Int b = *bp;
        
        while (ap < aEnd && bp < bEnd) {
            
            if (loadA) {
                a = *ap;
                loadA = false;
            }
            if (loadB) {
                b = *bp;
                loadB = false;
            }

          if (a == b) {
              ++count;
              ++ap;
              ++bp;
              loadA = true;
              loadB = true;
          }
          else if (a < b){
              ++ap;
              loadA = true;
          }
          else {
              ++bp;
              loadB = true;
          }
      }
    }
    return count;
}


/*! Count triangles

Use one thread per edge to count triangles.
Compare sorted neighbor lists linearly.
*/
__global__ static void kernel_tc(
    uint64_t * __restrict__ triangleCounts, //!< per-edge triangle counts
    const Int *const edgeSrc, //!< node ids for edge srcs
    const Int *const edgeDst, //!< node ids for edge dsts
    const Int *const nodes, //!< source node offsets in edgeDst
    const size_t edgeOffset, //!< where in the edge list this function should begin counting
    const size_t numEdges //!< how many edges to count triangles for
    ){
     
    const Int gx = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (Int i = gx + edgeOffset; i < edgeOffset + numEdges; i += blockDim.x * gridDim.x) {

        // get the src and dst node for this edge
        const Int src = edgeSrc[i];
        const Int dst = edgeDst[i];

        const Int src_edge = nodes[src];
        const Int src_edge_end = nodes[src + 1];

        const Int dst_edge = nodes[dst];
        const Int dst_edge_end = nodes[dst + 1];

        size_t count = linear_intersection_count(&edgeDst[src_edge], &edgeDst[src_edge_end], &edgeDst[dst_edge], &edgeDst[dst_edge_end]);

        triangleCounts[i] = count;
    }
}


// return 1 if search_val is in array between offets left and right, inclusive
__device__ static bool binary_search(const Int *const array, size_t left,
    size_t right, const Int search_val) {
    while (left <= right) {
        size_t mid = (left + right) / 2;
        Int val = array[mid];
        if (val < search_val) {
            left = mid + 1;
        } else if (val > search_val) {
            right = mid - 1;
        } else { // val == search_val
            return 1;
        }
    }
    return 0;
}

/*! Count triangles

Use one warp per edge to count triangles.
Compare neighbor lists in parallel with a binary search of the longer list

*/
template <size_t BLOCK_DIM_X>
__global__ static void
kernel_binary(
    uint64_t *__restrict__ edgeTriangleCounts, //<! per-edge triangle count
    const Int *edgeSrc,
    const Int *edgeDst,
    const Int *rowStarts, //<! offset in edgeSrc/edgeDst where each row starts
    const Int edgeOffset,
    const Int numEdges) {

  const size_t WARPS_PER_BLOCK = BLOCK_DIM_X / 32;
  static_assert(BLOCK_DIM_X % 32 ==
                0, "expect integer number of warps per block");
  typedef hipcub::WarpReduce<size_t> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage[WARPS_PER_BLOCK];


  const int warpIdx = threadIdx.x / 32; // which warp in thread block
  const int laneIdx = threadIdx.x % 32; // which thread in warp


  const size_t gwIdx = warpIdx + blockIdx.x * WARPS_PER_BLOCK;

  // one warp per edge
  for (Int edgeIdx = gwIdx; edgeIdx < numEdges; edgeIdx += WARPS_PER_BLOCK * gridDim.x) {

    size_t count = 0;

    // head and tail of edge
    const Int head = edgeSrc[edgeIdx];
    const Int tail = edgeDst[edgeIdx];

    // neighbor offsets for head of edge
    const Int headOffStart = rowStarts[head];
    const Int headOffEnd = rowStarts[head + 1];

    // neighbor offsets for tail of edge
    const Int tailOffStart = rowStarts[tail];
    const Int tailOffEnd = rowStarts[tail + 1];

    if (headOffEnd - headOffStart < tailOffEnd - tailOffStart) {
        for (const Int *u = &edgeDst[headOffStart] + laneIdx;
            u < &edgeDst[headOffEnd]; u += 32) {
        count +=
            binary_search(edgeDst, tailOffStart, tailOffEnd - 1, *u);
        }
    } else {
        for (const Int *u = &edgeDst[tailOffStart] + laneIdx;
            u < &edgeDst[tailOffEnd]; u += 32) {
        count +=
            binary_search(edgeDst, headOffStart, headOffEnd - 1, *u);
        }
    }



    size_t aggregate = WarpReduce(temp_storage[warpIdx]).Sum(count);

    if (laneIdx == 0) {
        edgeTriangleCounts[edgeIdx] = aggregate;
    }

  }
}




IMPACT2019TC::IMPACT2019TC(Config &c)  : CUDATriangleCounter(c) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(debug, "IMPACT 2019 TC, sizeof(Int) = {}", sizeof(Int));

    if (c.storage_ == "um") {
        GPUMemoryKind_ = GPUMemoryKind::Unified;
    } else if (c.storage_ == "zc") {
        GPUMemoryKind_ = GPUMemoryKind::ZeroCopy;
    } else {
        LOG(critical, "unknown gpu storage kind \"{}\"", c.storage_);
        exit(-1);
    }

    if ("linear" == c.kernel_) {
        kernelKind_ = KernelKind::Linear;
    } else if ("binary" == c.kernel_) {
        kernelKind_ = KernelKind::Binary;
    } else {
        LOG(critical, "unknown kernel kind \"{}\", expecting linear|binary", c.kernel_);
        exit(-1);
    }

    unifiedMemoryHints_ = c.hints_;
    nvtxRangePop();
}

IMPACT2019TC::~IMPACT2019TC() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    switch (GPUMemoryKind_) {
        case GPUMemoryKind::Unified: {
            CUDA_RUNTIME(hipFree(edgeSrc_d_));
            CUDA_RUNTIME(hipFree(edgeDst_d_));
            CUDA_RUNTIME(hipFree(cols_d_));
            CUDA_RUNTIME(hipFree(triangleCounts_));
            break;
        }
        case GPUMemoryKind::ZeroCopy: {
            CUDA_RUNTIME(hipHostUnregister(hostDAG_.edgeSrc_.data()));
            CUDA_RUNTIME(hipHostUnregister(hostDAG_.edgeDst_.data()));
            CUDA_RUNTIME(hipHostUnregister(hostDAG_.nodes_.data()));
            CUDA_RUNTIME(hipHostFree(triangleCounts_));   
            break;         
        }
        default:
            LOG(error, "unexpected GPUMemoryKind in dtor");
    } 
    nvtxRangePop();
}

void IMPACT2019TC::read_data(const std::string &path) {
    nvtxRangePush(__PRETTY_FUNCTION__);
    LOG(info, "reading {}", path);
    auto *reader = pangolin::EdgeListReader::from_file(path);
    auto edgeList = reader->read();
    if (edgeList.size() == 0) {
        LOG(warn, "empty edge list");
    }
    LOG(debug, "building DAG");
    hostDAG_ = DAG2019::from_edgelist(edgeList);

    LOG(info, "{} nodes", hostDAG_.num_nodes());
    LOG(info, "{} edges", hostDAG_.num_edges());
    nvtxRangePop();
}

void IMPACT2019TC::setup_data() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t edgeBytes = hostDAG_.edgeSrc_.size() * sizeof(Int);
    const size_t nodeBytes = hostDAG_.nodes_.size() * sizeof(Int);
    const size_t countBytes = hostDAG_.num_edges() * sizeof(*triangleCounts_);

    switch (GPUMemoryKind_) {
        case GPUMemoryKind::Unified: {
            CUDA_RUNTIME(hipMallocManaged(&edgeSrc_d_, edgeBytes));
            CUDA_RUNTIME(hipMallocManaged(&edgeDst_d_, edgeBytes));
            CUDA_RUNTIME(hipMallocManaged(&cols_d_, nodeBytes));
            CUDA_RUNTIME(hipMallocManaged(&triangleCounts_, countBytes));

            LOG(debug, "copying to unified memory");
            CUDA_RUNTIME(hipMemcpy(edgeSrc_d_, hostDAG_.edgeSrc_.data(), edgeBytes, hipMemcpyDefault));
            CUDA_RUNTIME(hipMemcpy(edgeDst_d_, hostDAG_.edgeDst_.data(), edgeBytes, hipMemcpyDefault)); 
            CUDA_RUNTIME(hipMemcpy(cols_d_, hostDAG_.nodes_.data(), nodeBytes, hipMemcpyDefault));

            // processor id is ignored
            if (unifiedMemoryHints_) {
                LOG(info, "using unified memory hints");
                CUDA_RUNTIME(hipMemAdvise(edgeSrc_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
                CUDA_RUNTIME(hipMemAdvise(edgeDst_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
                CUDA_RUNTIME(hipMemAdvise(cols_d_, nodeBytes, hipMemAdviseSetReadMostly, 0));
                for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
                    if (cudaDeviceProps_[i].concurrentManagedAccess) {
                        CUDA_RUNTIME(hipMemAdvise(edgeSrc_d_, edgeBytes, hipMemAdviseSetAccessedBy, i));
                        CUDA_RUNTIME(hipMemAdvise(edgeDst_d_, edgeBytes, hipMemAdviseSetAccessedBy, i));
                        CUDA_RUNTIME(hipMemAdvise(cols_d_, nodeBytes, hipMemAdviseSetAccessedBy, i));
                    } else {
                        LOG(warn, "skipping hipMemAdviseSetAccessedBy for device {}: hipDeviceProp_t.concurrentManagedAccess = 0", i);
                    }
                }
            }

            break;
        }
        case GPUMemoryKind::ZeroCopy: {
            // map host memory
            CUDA_RUNTIME(hipHostRegister(hostDAG_.edgeSrc_.data(), edgeBytes, hipHostRegisterMapped | hipHostRegisterPortable));
            CUDA_RUNTIME(hipHostRegister(hostDAG_.edgeDst_.data(), edgeBytes, hipHostRegisterMapped | hipHostRegisterPortable));
            CUDA_RUNTIME(hipHostRegister(hostDAG_.nodes_.data(), nodeBytes, hipHostRegisterMapped | hipHostRegisterPortable));

            // get valid device pointer
            CUDA_RUNTIME(hipHostGetDevicePointer(&edgeSrc_d_, hostDAG_.edgeSrc_.data(), 0));
            CUDA_RUNTIME(hipHostGetDevicePointer(&edgeDst_d_, hostDAG_.edgeDst_.data(), 0)); 
            CUDA_RUNTIME(hipHostGetDevicePointer(&cols_d_, hostDAG_.nodes_.data(), 0));

            // allocate memory for output
            CUDA_RUNTIME(hipHostAlloc(&triangleCounts_, countBytes, hipHostMallocMapped));
            break;
        }
        default: {
            LOG(critical, "unhandled value for gpu memory kind");
            exit(-1);
        }
    }

    nvtxRangePop();
}

size_t IMPACT2019TC::count() {
    nvtxRangePush(__PRETTY_FUNCTION__);
    const size_t numDev = gpus_.size();

    // split edges among devices
    size_t edgesPerDevice = (hostDAG_.num_edges() + numDev - 1) / numDev;
    LOG(debug, "{} edges per GPU", edgesPerDevice);

    size_t edgeOffset = 0;
    for (int i : gpus_) {
        CUDA_RUNTIME(hipSetDevice(i));

        size_t edgeCount = std::min(edgesPerDevice, hostDAG_.num_edges() - edgeOffset);
        LOG(debug, "GPU {} edges {}+{}", i, edgeOffset, edgeCount);

        // Launch the correct kind of kernel
        switch (kernelKind_) {
            case KernelKind::Linear: {
                LOG(debug, "linear kernel");
                dim3 dimBlock(256);
                size_t desiredGridSize = (edgeCount + dimBlock.x - 1) / dimBlock.x;
                dim3 dimGrid(std::min(size_t(std::numeric_limits<int>::max()), desiredGridSize));
                LOG(debug, "kernel dims {} x {}", dimGrid.x, dimBlock.x);
                kernel_tc<<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_, cols_d_, edgeOffset, edgeCount);
                break;
            }
            case KernelKind::Binary: {
                LOG(debug, "binary kernel");
                constexpr int dimBlock = 512;
                static_assert(dimBlock % 32 == 0, "Expect integer warps per block");
                const int warpsPerBlock = dimBlock / 32;
                size_t desiredGridSize = (edgeCount + warpsPerBlock - 1) / warpsPerBlock;
                dim3 dimGrid(std::min(size_t(std::numeric_limits<int>::max()), desiredGridSize));
                LOG(debug, "kernel dims {} x {}", dimGrid.x, dimBlock);
                kernel_binary<dimBlock><<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_, cols_d_, edgeOffset, edgeCount);
                break;
            }
            default: {
                LOG(critical, "unexpected kernelKind_");
                exit(-1);
            }
        }


        edgeOffset += edgesPerDevice;
    }
    
    for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
        CUDA_RUNTIME(hipSetDevice(i));
        LOG(debug, "Waiting for GPU {}", i);
        CUDA_RUNTIME(hipDeviceSynchronize());
    }

    nvtxRangePush("final reduction");
    auto start = std::chrono::system_clock::now();
    size_t total = 0;
    for(size_t i = 0; i < hostDAG_.num_edges(); ++i) {
        total += triangleCounts_[i];
    }
    auto elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(debug, "CPU reduction {}s", elapsed);

    nvtxRangePop();
    return total;
}
