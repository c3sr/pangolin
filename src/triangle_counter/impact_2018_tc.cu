#include "hip/hip_runtime.h"
#include "pangolin/logger.hpp"
#include "pangolin/reader/gc_tsv_reader.hpp"
#include "pangolin/triangle_counter/impact_2018_tc.hpp"
#include "pangolin/utilities.hpp"

#include <limits>
#include <nvToolsExt.h>

namespace pangolin {

__global__ static void kernel_tc(size_t *triangleCounts, Int *edgeSrc,
                                 Int *edgeDst, Int *nodes, size_t edgeOffset,
                                 size_t numEdges) {

  const Int gx = blockIdx.x * blockDim.x + threadIdx.x;

  for (Int i = gx + edgeOffset; i < edgeOffset + numEdges;
       i += blockDim.x * gridDim.x) {

    // get the src and dst node for this edge
    const Int src = edgeSrc[i];
    const Int dst = edgeDst[i];

    const Int srcEdge = nodes[src];
    const Int srcEdgeEnd = nodes[src + 1];

    const Int dstEdge = nodes[dst];
    const Int dstEdgeEng = nodes[dst + 1];

    size_t count = 0;

    Int *srcPtr = &edgeDst[srcEdge];
    Int *dstPtr = &edgeDst[dstEdge];

    while (srcPtr < &edgeDst[srcEdgeEnd] && dstPtr < &edgeDst[dstEdgeEng]) {

      const Int srcNbr = *srcPtr; // neighbor vertex of edge src
      const Int dstNbr = *dstPtr; // neighbor vertex of edge dst

      if (srcNbr == dstNbr) {
        ++count;
        ++srcPtr;
        ++dstPtr;
      } else if (srcNbr < dstNbr) {
        ++srcPtr;
      } else {
        ++dstPtr;
      }
    }

    triangleCounts[i] = count;
  }
}

IMPACT2018TC::IMPACT2018TC(Config &c) : CUDATriangleCounter(c) {
  nvtxRangePush(__PRETTY_FUNCTION__);
  SPDLOG_DEBUG(logger::console, "IMPACT 2018 TC, sizeof(Int) = {}",
               sizeof(Int));

  if (c.storage_ == "um") {
    GPUMemoryKind_ = GPUMemoryKind::Unified;
  } else if (c.storage_ == "zc") {
    GPUMemoryKind_ = GPUMemoryKind::ZeroCopy;
  } else {
    LOG(critical, "unknown gpu storage kind \"{}\"", c.storage_);
    exit(-1);
  }

  unifiedMemoryHints_ = c.hints_;

  nvtxRangePop();
}

IMPACT2018TC::~IMPACT2018TC() {
  nvtxRangePush(__PRETTY_FUNCTION__);
  switch (GPUMemoryKind_) {
  case GPUMemoryKind::Unified: {
    CUDA_RUNTIME(hipFree(edgeSrc_d_));
    CUDA_RUNTIME(hipFree(edgeDst_d_));
    CUDA_RUNTIME(hipFree(nodes_d_));
    CUDA_RUNTIME(hipFree(triangleCounts_));
    break;
  }
  case GPUMemoryKind::ZeroCopy: {
    CUDA_RUNTIME(hipHostUnregister(hostDAG_.edgeSrc_.data()));
    CUDA_RUNTIME(hipHostUnregister(hostDAG_.edgeDst_.data()));
    CUDA_RUNTIME(hipHostUnregister(hostDAG_.nodes_.data()));
    CUDA_RUNTIME(hipHostFree(triangleCounts_));
    break;
  }
  default:
    LOG(error, "unexpected GPUMemoryKind in dtor");
  }
  nvtxRangePop();
}

void IMPACT2018TC::read_data(const std::string &path) {
  nvtxRangePush(__PRETTY_FUNCTION__);
  LOG(info, "reading {}", path);
  pangolin::GraphChallengeTSVReader reader(path);
  auto edgeList = reader.read_edges();
  SPDLOG_DEBUG(logger::console, "building DAG");
  hostDAG_ = DAG2019::from_edgelist(edgeList);

  LOG(info, "{} nodes", hostDAG_.num_nodes());
  LOG(info, "{} edges", hostDAG_.num_edges());
  nvtxRangePop();
}

void IMPACT2018TC::setup_data() {

  nvtxRangePush(__PRETTY_FUNCTION__);
  const size_t edgeBytes = hostDAG_.edgeSrc_.size() * sizeof(Int);
  const size_t nodeBytes = hostDAG_.nodes_.size() * sizeof(Int);
  const size_t countBytes = hostDAG_.num_edges() * sizeof(*triangleCounts_);

  switch (GPUMemoryKind_) {
  case GPUMemoryKind::Unified: {
    CUDA_RUNTIME(hipMallocManaged(&edgeSrc_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&edgeDst_d_, edgeBytes));
    CUDA_RUNTIME(hipMallocManaged(&nodes_d_, nodeBytes));
    CUDA_RUNTIME(hipMallocManaged(&triangleCounts_, countBytes));

    SPDLOG_DEBUG(logger::console, "copying to unified memory");
    CUDA_RUNTIME(hipMemcpy(edgeSrc_d_, hostDAG_.edgeSrc_.data(), edgeBytes,
                            hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpy(edgeDst_d_, hostDAG_.edgeDst_.data(), edgeBytes,
                            hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpy(nodes_d_, hostDAG_.nodes_.data(), nodeBytes,
                            hipMemcpyDefault));

    // processor id is ignored
    if (unifiedMemoryHints_) {
      LOG(info, "using unified memory hints");
      CUDA_RUNTIME(
          hipMemAdvise(edgeSrc_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
      CUDA_RUNTIME(
          hipMemAdvise(edgeDst_d_, edgeBytes, hipMemAdviseSetReadMostly, 0));
      CUDA_RUNTIME(
          hipMemAdvise(nodes_d_, nodeBytes, hipMemAdviseSetReadMostly, 0));
      for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
        if (cudaDeviceProps_[i].concurrentManagedAccess) {
          CUDA_RUNTIME(hipMemAdvise(edgeSrc_d_, edgeBytes,
                                     hipMemAdviseSetAccessedBy, i));
          CUDA_RUNTIME(hipMemAdvise(edgeDst_d_, edgeBytes,
                                     hipMemAdviseSetAccessedBy, i));
          CUDA_RUNTIME(hipMemAdvise(nodes_d_, nodeBytes,
                                     hipMemAdviseSetAccessedBy, i));
        } else {
          LOG(warn,
              "skipping hipMemAdviseSetAccessedBy for device {}: "
              "hipDeviceProp_t.concurrentManagedAccess = 0",
              i);
        }
      }
    }

    break;
  }
  case GPUMemoryKind::ZeroCopy: {
    // map host memory
    CUDA_RUNTIME(
        hipHostRegister(hostDAG_.edgeSrc_.data(), edgeBytes,
                         hipHostRegisterMapped | hipHostRegisterPortable));
    CUDA_RUNTIME(
        hipHostRegister(hostDAG_.edgeDst_.data(), edgeBytes,
                         hipHostRegisterMapped | hipHostRegisterPortable));
    CUDA_RUNTIME(
        hipHostRegister(hostDAG_.nodes_.data(), nodeBytes,
                         hipHostRegisterMapped | hipHostRegisterPortable));

    // get valid device pointer
    CUDA_RUNTIME(
        hipHostGetDevicePointer(&edgeSrc_d_, hostDAG_.edgeSrc_.data(), 0));
    CUDA_RUNTIME(
        hipHostGetDevicePointer(&edgeDst_d_, hostDAG_.edgeDst_.data(), 0));
    CUDA_RUNTIME(
        hipHostGetDevicePointer(&nodes_d_, hostDAG_.nodes_.data(), 0));

    // allocate memory for output
    CUDA_RUNTIME(
        hipHostAlloc(&triangleCounts_, countBytes, hipHostMallocMapped));
    break;
  }
  default: {
    LOG(critical, "unhandled value for gpu memory kind");
    exit(-1);
  }
  }

  nvtxRangePop();
}

size_t IMPACT2018TC::count() {
  nvtxRangePush(__PRETTY_FUNCTION__);
  const size_t numDev = gpus_.size();

  // split edges among devices
  size_t edgesPerDevice = (hostDAG_.num_edges() + numDev - 1) / numDev;
  SPDLOG_DEBUG(logger::console, "{} edges per GPU", edgesPerDevice);

  size_t edgeOffset = 0;
  for (int i : gpus_) {
    CUDA_RUNTIME(hipSetDevice(i));

    size_t edgeCount =
        std::min(edgesPerDevice, hostDAG_.num_edges() - edgeOffset);
    SPDLOG_DEBUG(logger::console, "GPU {} edges {}+{}", i, edgeOffset,
                 edgeCount);

    dim3 dimBlock(256);
    size_t desiredGridSize = (edgeCount + dimBlock.x - 1) / dimBlock.x;
    dim3 dimGrid(
        std::min(size_t(std::numeric_limits<int>::max()), desiredGridSize));

    SPDLOG_DEBUG(logger::console, "kernel dims {} x {}", dimGrid.x, dimBlock.x);
    kernel_tc<<<dimGrid, dimBlock>>>(triangleCounts_, edgeSrc_d_, edgeDst_d_,
                                     nodes_d_, edgeOffset, edgeCount);
    edgeOffset += edgesPerDevice;
  }

  for (int i : std::set<int>(gpus_.begin(), gpus_.end())) {
    CUDA_RUNTIME(hipSetDevice(i));
    SPDLOG_DEBUG(logger::console, "Waiting for GPU {}", i);
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  nvtxRangePush("final reduction");
  auto start = std::chrono::system_clock::now();
  size_t total = 0;
  for (size_t i = 0; i < hostDAG_.num_edges(); ++i) {
    total += triangleCounts_[i];
  }
  auto elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  nvtxRangePop();
  SPDLOG_DEBUG(logger::console, "CPU reduction {}s", elapsed);

  nvtxRangePop();
  return total;
}

} // namespace pangolin